#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

// Define and implement the GPU addition function
// This version is a vector addition, with N threads
// and and N blocks
// Adding one a and b instance and storing in one c instance.
__global__ void add(int *a, int *b, int *c)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  c[index] = a[index] + b[index];
}

// Nmber of blocks
#define N (2048*2048)
#define THREADS_PER_BLOCK 512


int main()
{
  int *a, *b, *c; // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = N* sizeof(int);
  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  // Allocate memory for the host a, b, and c arrays
  a = (int*)malloc(size);
  b = (int*)malloc(size);
  c = (int*)malloc(size);
  // Store known values in the a and b arrays
  for (int i = 0; i < N; ++i)
    {
      a[i] = 10*i;
      b[i] = 20*i;
    }
  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  // Launch add() kernel on GPU with N threads on 1 block
  add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
  hipError_t err = hipGetLastError();  
  if(hipSuccess != err)
  {
    std::cout << "cuda kernel error: " << hipGetErrorString(err) << std::endl;
  }

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  // Print results removed...too big
  // for (int i = 0; i < N; ++i)
  //   {
  //     std::cout << "sum[" << i << "] is " << c[i] << std::endl;
  //   }
  std::cout << "Sum[0] is " << c[0] << std::endl;
  std::cout << "Sum[" << N - 1 << "] is " << c[N-1] << std::endl;
  // Cleanup
  free(a);
  free(b);
  free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
