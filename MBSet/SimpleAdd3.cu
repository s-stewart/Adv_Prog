#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

// Define and implement the GPU addition function
// This version is a vector addition, with N threads
// and one block.
// Adding one a and b instance and storing in one c instance.
__global__ void add(int *a, int *b, int *c)
{
  c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

// Nmber of blocks
#define N 512

int main()
{
  int *a, *b, *c; // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = N* sizeof(int);
  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  // Allocate memory for the host a, b, and c arrays
  a = (int*)malloc(size);
  b = (int*)malloc(size);
  c = (int*)malloc(size);
  // Store known values in the a and b arrays
  for (int i = 0; i < N; ++i)
    {
      a[i] = 10*i;
      b[i] = 20*i;
    }
  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  // Launch add() kernel on GPU with N threads on 1 block
  add<<<1,N>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  // Print results
  for (int i = 0; i < N; ++i)
    {
      std::cout << "sum[" << i << "] is " << c[i] << std::endl;
    }
  // Cleanup
  free(a);
  free(b);
  free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
